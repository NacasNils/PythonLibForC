#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// stb_import
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image/stb_image_write.h"

#define RED 0
#define GREEN 1
#define BLUE 2

extern "C" {
    int remove_color(const char *input, int mode, const char *o_dir);
}


// CUDA Error checking macro
#define CUDA_CHECK(call) \
    { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            printf("CUDA error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    }

// CUDA Kernel for grayscale conversion
__global__ void grayscaleKernel(unsigned char *img, unsigned char *gray_img, int width, int height, int channels) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_pixels = width * height;
    
    if (idx < total_pixels) {
        int i = idx * channels;
        gray_img[idx] = (unsigned char)((img[i] + img[i + 1] + img[i + 2]) / 3);
    }
}

// CUDA Kernel to remove a specific color
__global__ void removeColorKernel(unsigned char *img, int width, int height, int channels, int color) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_pixels = width * height;

    if (idx < total_pixels) {
        img[idx * channels + color] = 0;  // Set the specified color channel to 0
    }
}

// Function to handle grayscale conversion using CUDA
int remove_all(int width, int height, int channels, unsigned char *img, const char *o_dir) {
    int img_size = width * height * channels;
    int gray_img_size = width * height;
    
    unsigned char *d_img, *d_gray_img;
    unsigned char *gray_img = (unsigned char *)malloc(gray_img_size);

    if (!gray_img) return -2;

    // Allocate device memory
    CUDA_CHECK(hipMalloc((void **)&d_img, img_size));
    CUDA_CHECK(hipMalloc((void **)&d_gray_img, gray_img_size));

    // Copy input image to device
    CUDA_CHECK(hipMemcpy(d_img, img, img_size, hipMemcpyHostToDevice));

    // Launch CUDA kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (width * height + threadsPerBlock - 1) / threadsPerBlock;
    grayscaleKernel<<<blocksPerGrid, threadsPerBlock>>>(d_img, d_gray_img, width, height, channels);
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy result back to host
    CUDA_CHECK(hipMemcpy(gray_img, d_gray_img, gray_img_size, hipMemcpyDeviceToHost));

    // Save output image
    char output[256];
    sprintf(output, "./%s/out_gray.png", o_dir);
    stbi_write_png(output, width, height, 1, gray_img, width);

    // Free memory
    hipFree(d_img);
    hipFree(d_gray_img);
    free(gray_img);

    return 1;
}

// Function to remove a color channel using CUDA
int remove_rgb(int width, int height, int channels, unsigned char *img, int color, const char *o_dir) {
    int img_size = width * height * channels;
    
    unsigned char *d_img;
    CUDA_CHECK(hipMalloc((void **)&d_img, img_size));
    CUDA_CHECK(hipMemcpy(d_img, img, img_size, hipMemcpyHostToDevice));

    // Launch CUDA kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (width * height + threadsPerBlock - 1) / threadsPerBlock;
    removeColorKernel<<<blocksPerGrid, threadsPerBlock>>>(d_img, width, height, channels, color);
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy result back to host
    CUDA_CHECK(hipMemcpy(img, d_img, img_size, hipMemcpyDeviceToHost));

    // Save output image
    char output[256];
    sprintf(output, "./%s/out_rgb.png", o_dir);
    stbi_write_png(output, width, height, channels, img, width * channels);

    // Free memory
    hipFree(d_img);

    return 1;
}

// Main function to process image
int remove_color(const char *input, int mode, const char *o_dir) {
    int width, height, channels;
    unsigned char *img = stbi_load(input, &width, &height, &channels, 0);

    if (img == NULL) return -1;

    int result;
    if (mode == 0) {
        result = remove_all(width, height, channels, img, o_dir);
    } else if (mode == 1) {
        result = remove_rgb(width, height, channels, img, RED, o_dir);
    } else if (mode == 2) {
        result = remove_rgb(width, height, channels, img, GREEN, o_dir);
    } else if (mode == 3) {
        result = remove_rgb(width, height, channels, img, BLUE, o_dir);
    } else {
        result = 2;
    }

    stbi_image_free(img);
    return result;
}

// Main function for testing
int main(int argc, char *argv[]) {
    if (argc != 4) {
        printf("Usage: %s <input_image> <mode> <output_dir>\n", argv[0]);
        return -1;
    }

    const char *input = argv[1];
    int mode = atoi(argv[2]);
    const char *o_dir = argv[3];

    int status = remove_color(input, mode, o_dir);
    if (status == 1) {
        printf("Image processed successfully.\n");
    } else {
        printf("Error processing image. Code: %d\n", status);
    }

    return 0;
}

